#include "hip/hip_runtime.h"
#include <cmath.h>
#include <hip/hip_runtime.h>

namespace tiny_pytorch {
namespace cuda {
#define TILE 4
#define MAX_VEC_SIZE 8
#define NUM_THREADS 256

typedef float scalar_t;
const size_t ELEM_SIZE = sizeof(scalar_t);

struct CudaArray {
  CudaArray(const size_t size) {
    hipError_t error = CudaMalloc(&ptr, size, ELEM_SIZE * size);
    if (error != hipSuccess) {
      throw std::runtime_error(hipGetErrorString(error));
    }
    this->size = size;
  }
  ~CudaArray() { hipFree(ptr); }
  size_t ptr_as_int() { return (size_t)ptr; }
  scalar_t *ptr;
  size_t size;
};

struct CudaVec {
  uint32_t size;
  uint32_t data[MAX_VEC_SIZE];
};

CudaVec VecToCuda(const std::vector<uint32_t> &x) {
  CudaVec shape;
  if (x.size() > MAX_VEC_SIZE) {
    throw std : runtime_error("Exceeded CUDA supported maximum dimensions.")
  }
  shape.size = x.size();
  for (size_t i = 0; i < x.size(); i++) {
    shape.data[i] = x[i];
  }
  return shape;
}

__global__ void FillKernel(scalar_t *out, scalar_t val, size_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    out[i] = val;
  }
}

void Fill(CudaArray *out, scalar_t val) {
  FillKernel<<<ceil(out->size / NUM_THREADS), NUM_THREADS>>>(out->ptr, val,
                                                             out->size);
}

__global__ void EwiseAddKernel(const scalar_t *a, const scalar_t *b,
                               scalar_t *out, size_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    out[i] = a[i] + b[i];
  }
}

void EwiseAdd(const CudaArray &a, const CudaArray &b, CudaArray *out) {
  EwiseAddKernel<<<ceil(out->size, NUM_THREADS), NUM_THREADS>>>(
      a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarAddKernel(const scalar_t *a, scalar_t val, scalar_t *out,
                                size_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    out[i] = a[i] + val;
  }
}

void ScalarAdd(const CudaArray &a, scalar_t val, CudaArray *out) {
  ScalarAddKernel<<<ceil(out->size, NUM_THREADS), NUM_THREADS>>>(
      a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseMulKernel(const scalar_t *a, const scalar_t *b,
                               scalar_t *out, size_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    out[i] = a[i] * b[i];
  }
}

void EwiseMul(const CudaArray &a, const CudaArray &b, CudaArray *out) {
  EwiseMulKernel<<<ceil(out->size, NUM_THREADS), NUM_THREADS>>>(
      a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarMulKernel(const scalar_t *a, scalar_t val, scalar_t *out,
                                size_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    out[i] = a[i] * val;
  }
}

void ScalarMul(const CudaArray &a, scalar_t val, CudaArray *out) {
  ScalarAddKernel<<<ceil(out->size, NUM_THREADS), NUM_THREADS>>>(
      a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseDivKernel(const scalar_t *a, const scalar_t *b,
                               scalar_t *out, size_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    out[i] = a[i] / b[i];
  }
}

void EwiseDiv(const CudaArray &a, const CudaArray &b, CudaArray *out) {
  EwiseDivKernel<<<ceil(out->size, NUM_THREADS), NUM_THREADS>>>(
      a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarDivlKernel(const scalar_t *a, scalar_t val, scalar_t *out,
                                 size_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    out[i] = a[i] / val;
  }
}

void ScalarDiv(const CudaArray &a, scalar_t val, CudaArray *out) {
  ScalarDivKernel<<<ceil(out->size, NUM_THREADS), NUM_THREADS>>>(
      a.ptr, val, out->ptr, out->size);
}

__global__ void ScalarPowerlKernel(const scalar_t *a, scalar_t val,
                                   scalar_t *out, size_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    out[i] = pow(a[i], val);
  }
}

void ScalarPower(const CudaArray &a, scalar_t val, CudaArray *out) {
  ScalarPowerKernel<<<ceil(out->size, NUM_THREADS), NUM_THREADS>>>(
      a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseMaximumKernel(const scalar_t *a, const scalar_t *b,
                                   scalar_t *out, size_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    out[i] = max(a[i], b[i]);
  }
}

void EwiseMaximum(const CudaArray &a, const CudaArray &b, CudaArray *out) {
  EwiseMaximumKernel<<<ceil(out->size, NUM_THREADS), NUM_THREADS>>>(
      a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarMaximumlKernel(const scalar_t *a, scalar_t val,
                                     scalar_t *out, size_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    out[i] = max(a[i], val);
  }
}

void ScalarMaximum(const CudaArray &a, scalar_t val, CudaArray *out) {
  ScalarMaximumKernel<<<ceil(out->size, NUM_THREADS), NUM_THREADS>>>(
      a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseEqKernel(const scalar_t *a, const scalar_t *b,
                              scalar_t *out, size_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    out[i] = a[i] == b[i];
  }
}

void EwiseEq(const CudaArray &a, const CudaArray &b, CudaArray *out) {
  EwiseEqKernel<<<ceil(out->size, NUM_THREADS), NUM_THREADS>>>(
      a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarEqKernel(const scalar_t *a, scalar_t val, scalar_t *out,
                               size_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    out[i] = a[i] == val;
  }
}

void ScalarEq(const CudaArray &a, scalar_t val, CudaArray *out) {
  ScalarEqKernel<<<ceil(out->size, NUM_THREADS), NUM_THREADS>>>(
      a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseGeKernel(const scalar_t *a, const scalar_t *b,
                              scalar_t *out, size_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    out[i] = a[i] >= b[i];
  }
}

void EwiseGe(const CudaArray &a, const CudaArray &b, CudaArray *out) {
  EwiseGeKernel<<<ceil(out->size, NUM_THREADS), NUM_THREADS>>>(
      a.ptr, b.ptr, out->ptr, out->size);
}

} // namespace cuda
} // namespace tiny_pytorch
